#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>

#define BLOCK_SIZE 256 // Number of threads in each block

/**********************************************************
 * Checks the error between numerical and exact solutions
 **********************************************************/
double checkSolution(double xStart, double yStart,
                     int maxXCount, int maxYCount,
                     double *u,
                     double deltaX, double deltaY,
                     double alpha, double *fx_thing, double *fy_thing)
{
#define U(XX,YY) u[(YY)*maxXCount+(XX)]
    int x, y;
    double localError, error = 0.0;

    for (y = 1; y < (maxYCount-1); y++)
    {
        for (x = 1; x < (maxXCount-1); x++)
        {
            localError = U(x,y) - fx_thing[x-1]*fy_thing[y-1];
            error += localError*localError;
        }
    }
    return sqrt(error)/((maxXCount-2)*(maxYCount-2));
}

__global__ void jacobiIteration(int n, int m, double alpha, double relax, double cx, double cy, double cc, double *u, double *u_old, double *fx_thing, double *fy_thing) {
    #define SRC(XX,YY) u_old[(YY)*(n+2)+(XX)]
    #define DST(XX,YY) u[(YY)*(n+2)+(XX)]
    int x = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int y = blockIdx.y * blockDim.y + threadIdx.y + 1;
    if (x < n+1 && y < n+1) {
        double f = -alpha*fx_thing[x-1]*fy_thing[y-1] - 2.0*fx_thing[x-1] - 2.0*fy_thing[y-1];
        double updateVal = (	(SRC(x-1,y) + SRC(x+1,y))*cx +
                        (SRC(x,y-1) + SRC(x,y+1))*cy +
                        SRC(x,y)*cc - f
                    )/cc;
        DST(x,y) = SRC(x,y) - relax*updateVal;
    }
}


int main(int argc, char **argv)
{
    int n, m, mits;
    double alpha, tol, relax;
    // double maxAcceptableError;
    // double error;
    int allocCount;
    int iterationCount, maxIterationCount;
    // double t1, t2;

//    printf("Input n,m - grid dimension in x,y direction:\n");
    scanf("%d,%d", &n, &m);
//    printf("Input alpha - Helmholtz constant:\n");
    scanf("%lf", &alpha);
//    printf("Input relax - successive over-relaxation parameter:\n");
    scanf("%lf", &relax);
//    printf("Input tol - error tolerance for the iterrative solver:\n");
    scanf("%lf", &tol);
//    printf("Input mits - maximum solver iterations:\n");
    scanf("%d", &mits);

    allocCount = (n+2)*(m+2);

    // Solve in [-1, 1] x [-1, 1]
    double xLeft = -1.0, xRight = 1.0;
    double yBottom = -1.0, yUp = 1.0;

    double deltaX = (xRight-xLeft)/(n-1);
    double deltaY = (yUp-yBottom)/(m-1);

    double cx = 1.0/(deltaX*deltaX);
    double cy = 1.0/(deltaY*deltaY);
    double cc = -2.0*cx-2.0*cy-alpha;

    // Allocate memory for pre-calculated stuff for the GPU
    double *fx_thing, *fy_thing;
    hipMallocManaged(&fx_thing, n*sizeof(double));
    hipMallocManaged(&fy_thing, m*sizeof(double));

    // Precalucate stuff to save time

    int x,y;
    for (x = 1; x < n+1; x++) {
        double fX = xLeft + (x-1)*deltaX;
        fx_thing[x-1] = 1.0-fX*fX;
    }

    for (y = 1; y < m+1; y++) {
        double fY = yBottom + (y-1)*deltaY;
        fy_thing[y-1] = 1.0-fY*fY;
    }

    // Allocate u and u_old for GPU
    double *d_u, *d_u_old;
    hipMalloc(&d_u, allocCount * sizeof(double));
    hipMalloc(&d_u_old, allocCount * sizeof(double));
    hipMemset(d_u, 0, allocCount * sizeof(double));
    hipMemset(d_u_old, 0, allocCount * sizeof(double));

    // Calculate GridSize
    dim3 gridSize(ceil(n/sqrt(BLOCK_SIZE)), ceil(m/sqrt(BLOCK_SIZE)));

    // Calculate Block size
    dim3 blockSize(sqrt(BLOCK_SIZE), sqrt(BLOCK_SIZE));

    printf("Grid size: %dx%d\n\n", n, m);

    //Run main loop
    iterationCount = 0;
    maxIterationCount = mits;
    double *tmp;
    // maxAcceptableError = tol;
    double t1 = clock();
    while (iterationCount < maxIterationCount) {
        jacobiIteration<<<gridSize, blockSize>>>(n, m, alpha, relax, cx, cy, cc, d_u, d_u_old, fx_thing, fy_thing);

        hipDeviceSynchronize();

        iterationCount++;

        // Swap the buffers
        tmp = d_u_old;
        d_u_old = d_u;
        d_u = tmp;
    }
    double t2 = clock();
    printf( "Iterations=%3d Elapsed time is %f\n", iterationCount, (double)(t2 - t1)/CLOCKS_PER_SEC);

    // Copy grid to host
    double *u_old = (double*)malloc(allocCount * sizeof(double));
    hipMemcpy(u_old, d_u_old, allocCount * sizeof(double), hipMemcpyDeviceToHost);

    // u_old holds the solution after the most recent buffers swap
    double absoluteError = checkSolution(xLeft, yBottom,
                                         n+2, m+2,
                                         u_old,
                                         deltaX, deltaY,
                                         alpha, fx_thing, fy_thing);
    printf("The error of the iterative solution is %g\n", absoluteError);
    

    // Free GPU memory
    hipFree(fy_thing);
    hipFree(fx_thing);
    hipFree(d_u_old);
    hipFree(d_u);

    // Free host memory
    free(u_old);
}